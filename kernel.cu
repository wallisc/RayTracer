#include "hip/hip_runtime.h"
#include <algorithm>
#include <stdio.h>
#include <float.h>

#include "Light.h"
#include "Camera.h"
#include "PointLight.h"
#include "Sphere.h"
#include "Box.h"
#include "Plane.h"
#include "Triangle.h"
#include "glm/glm.hpp"
#include "PhongShader.h"
#include "SmoothTriangle.h"
#include "CookTorranceShader.h"
#include "hipError_t.h"
#include "kernel.h"
#include "hiprand.h"

#define kNoShapeFound NULL

using glm::vec3;

const int kBlockWidth = 16;
const float kMaxDist = FLT_MAX;

__device__ bool isInShadow(const Ray &shadow, BVHTree *tree, float intersectParam) {
   return false;
   //for (int i = 0; i < geomCount; i++) {
   //   float dist = geomList[i]->getIntersection(shadow);
   //   if (isFloatAboveZero(dist) && isFloatLessThan(dist, intersectParam)) { 
   //      return true;
   //   }
   //}
   //return false;
}

// Find the closest shape. The index of the intersecting object is stored in
// retOjIdx and the t-value along the input ray is stored in retParam
//
// If no intersection is found, retObjIdx is set to 'kNoShapeFound'
__device__ void getClosestIntersection(const Ray &ray, BVHTree *tree, 
      Geometry **retObj, float *retParam) {
   float t = kMaxDist;
   Geometry *closestGeom = kNoShapeFound;

   BVHNode *stack[kMaxStackSize];
   int stackSize = 0;
   bool justPoppedStack = false;

   BVHNode *cursor = tree->root;
     
   do {
      if (stackSize >= kMaxStackSize) {
         printf("Stack full, aborting!\n");
         return;
      }
         
      // If at a leaf
      if (cursor->geom) {
         float dist = cursor->geom->getIntersection(ray);
         //If two shapes are overlapping, pick the one with the closest facing normal
         if (isFloatEqual(t, dist)) {
            glm::vec3 oldNorm = closestGeom->getNormalAt(ray, t);
            glm::vec3 newNorm = cursor->geom->getNormalAt(ray, dist);
            glm::vec3 eye = glm::normalize(-ray.d);
            float newDot = glm::dot(eye, newNorm);
            float oldDot = glm::dot(eye, oldNorm);
            if (newDot > oldDot) {
               closestGeom = cursor->geom;
               t = dist;
            }
         // Otherwise, if one face is front of the current one
         } else if (dist < t && isFloatAboveZero(dist)) {
            t = dist;
            closestGeom = cursor->geom;
         }
      } else if (!justPoppedStack && isFloatAboveZero(cursor->left->bb.getIntersection(ray)) && cursor->left->bb.getIntersection(ray) < t) {
         //go left
         stack[stackSize++] = cursor;
         cursor = cursor->left;
         justPoppedStack = false;
         continue;
      } else if (cursor->right && isFloatAboveZero(cursor->right->bb.getIntersection(ray)) && cursor->right->bb.getIntersection(ray) < t) {
         //go right
         cursor = cursor->right;
         justPoppedStack = false;
         continue;
      }

      if(stackSize > 0) {
         // Pop the stack
         cursor = stack[stackSize - 1]; 
         justPoppedStack = true;
      }
      stackSize--;
   } while(stackSize >= 0);

   for (int planeIdx = 0; planeIdx < tree->planeListSize; planeIdx++) {
      float dist = tree->planeList[planeIdx]->getIntersection(ray);
      if (isFloatLessThan(dist, t) && isFloatAboveZero(dist)) {
         closestGeom = tree->planeList[planeIdx];
         t = dist;
      }

   }

   *retObj = closestGeom;
   *retParam = t;
}

template <int invRecLevel>
__device__ glm::vec3 getReflection(glm::vec3 point, glm::vec3 normal, glm::vec3 eyeVec, 
   BVHTree *tree, Light *lights[], int lightCount, Shader **shader) {

   Ray reflectRay(point, 2.0f * glm::dot(normal, eyeVec) * normal - eyeVec);
   reflectRay.o += BIG_EPSILON * reflectRay.d;
   Geometry *closestGeom;
   float t;

   getClosestIntersection(reflectRay, tree, &closestGeom, &t);
   if (closestGeom != kNoShapeFound) {
      return shadeObject<invRecLevel>(tree, 
            lights, lightCount,
            closestGeom, t,
            reflectRay, shader);
   } 
   return vec3(0.0f);
}

template <>
__device__ glm::vec3 getReflection<0>(glm::vec3 point, glm::vec3 normal, glm::vec3 eyeVec, 
   BVHTree *tree, Light *lights[], int lightCount, 
   Shader **shader) { return vec3(0.0f); }

template <int invRecLevel>
__device__ glm::vec3 getRefraction(glm::vec3 point, glm::vec3 normal, float ior, glm::vec3 eyeVec, 
   BVHTree *tree, Light *lights[], int lightCount, Shader **shader) {

   float n1, n2;
   vec3 refrNorm;
   vec3 d = -eyeVec;

   if (isFloatLessThan(glm::dot(eyeVec, normal), 0.0f)) {
      n1 = ior; n2 = kAirIOR;
      refrNorm = -normal;
   } else { 
      n1 = kAirIOR; n2 = ior;
      refrNorm = normal;
   }

   float dDotN = glm::dot(d, refrNorm);
   float nr = n1 / n2;
   float discriminant = 1.0f - nr * nr * (1.0f - dDotN * dDotN);
   if (discriminant > 0.0f) {
      vec3 refracDir = nr * (d - refrNorm * dDotN) - refrNorm * sqrtf(discriminant);
      Ray refracRay(point, refracDir);
      refracRay.o += BIG_EPSILON * refracRay.d;
      Geometry *closestGeom;
      float t;
      getClosestIntersection(refracRay, tree, &closestGeom, &t);
      if (closestGeom != kNoShapeFound) {
         return shadeObject<invRecLevel>(tree,
               lights, lightCount,
               closestGeom, t,
               refracRay, shader);
      }
   } 
   return vec3(0.0f);
}

template <>
__device__ glm::vec3 getRefraction<0>(glm::vec3 point, glm::vec3 normal, float ior, glm::vec3 eyeVec, 
   BVHTree *tree, Light *lights[], int lightCount, 
   Shader **shader) { return vec3(0.0f); }


//Note: The ray parameter must stay as a copy (not a reference) 
template <int invRecLevel> 
__device__ vec3 shadeObject(BVHTree *tree, 
      Light *lights[], int lightCount, Geometry* geom, 
      float intParam, Ray ray, Shader **shader) {

   glm::vec3 intersectPoint = ray.getPoint(intParam);
   Material m = geom->getMaterial();
   vec3 normal = geom->getNormalAt(ray, intParam);
   vec3 eyeVec = glm::normalize(-ray.d);
   vec3 totalLight(0.0f);

   for(int lightIdx = 0; lightIdx < lightCount; lightIdx++) {
      vec3 light = lights[lightIdx]->getLightAtPoint(geom, intersectPoint);
      vec3 lightDir = lights[lightIdx]->getLightDir(intersectPoint);
      Ray shadow = lights[lightIdx]->getShadowFeeler(intersectPoint);
      float intersectParam = geom->getIntersection(shadow);
      bool inShadow = isInShadow(shadow, tree, intersectParam); 

      totalLight += (*shader)->shade(m.clr, m.amb, m.dif, m.spec, m.rough, 
            eyeVec, lightDir, light, normal, 
            inShadow);
   }

   vec3 reflectedLight(0.0f);
   if (m.refl > 0.0f && invRecLevel - 1 > 0) {
      reflectedLight = getReflection<invRecLevel - 1>(intersectPoint, 
         normal, eyeVec, tree, lights, lightCount, shader);
   }

   vec3 refractedLight(0.0f);
   if (m.refr > 0.0f && invRecLevel - 1 > 0) {
      refractedLight = getRefraction<invRecLevel - 1>(intersectPoint, 
         normal, m.ior, eyeVec, tree, lights, lightCount, shader);

   }

   return totalLight * (1.0f - m.refl - m.alpha)
      + m.refl * reflectedLight+ m.alpha * refractedLight;
}

template <> 
__device__ vec3 shadeObject<0>(BVHTree *tree, 
      Light *lights[], int lightCount, int objIdx, 
      float intParam, Ray ray, Shader **shader) { return vec3(0.0f); }

__global__ void initScene(Geometry *geomList[], Plane *planeList[], Light *lights[], TKSphere *sphereTks, int numSpheres,
      TKPlane *planeTks, int numPlanes, TKTriangle *triangleTks, int numTris, TKBox *boxTks, int numBoxes,
      TKSmoothTriangle *smthTriTks, int numSmthTris, TKPointLight *pLightTks, int numPointLights, 
      Shader **shader, ShadingType stype) {
   int geomIdx = 0;
   int lightIdx = 0;

   // This should really only be run with one thread and block anyways, but this is a safety check
   if (blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0 && threadIdx.y == 0) {

      // Setup the shader
      switch(stype) {
      case PHONG:
         *shader = new PhongShader(); 
         break;
      case COOK_TORRANCE:
         *shader = new CookTorranceShader();
         break;
      default:
         printf("Improper shading type specified\n");
         break;
      }

      // Add all the geometry
      for (int i = 0; i < numSpheres; i++) {
         const TKSphere &s = sphereTks[i];
         const TKFinish f = s.mod.fin;
         Material m(s.mod.pig.clr, s.mod.pig.f, f.amb, f.dif, f.spec, f.rough, f.refl, f.refr, f.ior);
         geomList[geomIdx++] = new Sphere(s.p, s.r, m, s.mod.trans, s.mod.invTrans);
      }

      for (int i = 0; i < numPlanes; i++) {
         const TKPlane &p = planeTks[i];
         const TKFinish &f = p.mod.fin;
         Material m(p.mod.pig.clr, p.mod.pig.f, f.amb, f.dif, f.spec, f.rough, f.refl, f.refr, f.ior);
         planeList[i] = new Plane(p.d, p.n, m, p.mod.trans, p.mod.invTrans);
      }

      for (int i = 0; i < numTris; i++) {
         const TKTriangle &t = triangleTks[i];
         const TKFinish f = t.mod.fin;
         Material m(t.mod.pig.clr, t.mod.pig.f, f.amb, f.dif, f.spec, f.rough, f.refl, f.refr, f.ior);
         geomList[geomIdx++] = new Triangle(t.p1, t.p2, t.p3, m, t.mod.trans, 
               t.mod.invTrans);
      }

      for (int i = 0; i < numBoxes; i++) {
         const TKBox &b = boxTks[i];
         const TKFinish f = b.mod.fin;
         Material m(b.mod.pig.clr, b.mod.pig.f, f.amb, f.dif, f.spec, f.rough, f.refl, f.refr, f.ior);
         geomList[geomIdx++] = new Box(b.p1, b.p2, m, b.mod.trans, b.mod.invTrans);
      }

      for (int i = 0; i < numSmthTris; i++) {
         const TKSmoothTriangle &t = smthTriTks[i];
         const TKFinish f = t.mod.fin;
         Material m(t.mod.pig.clr, t.mod.pig.f, f.amb, f.dif, f.spec, f.rough, f.refl, f.refr, f.ior);
         geomList[geomIdx++] = new SmoothTriangle(t.p1, t.p2, t.p3, t.n1, t.n2, t.n3, 
               m, t.mod.trans, t.mod.invTrans);

      }

      // Add all the lights
      for (int i = 0; i < numPointLights; i++) {
         TKPointLight &p = pLightTks[i];
         lights[lightIdx++] = new PointLight(p.pos, p.clr);
      }
   }
}

typedef struct SortFrame {
   int size;
   Geometry **arr;
   __device__ SortFrame(int nSize = 0, Geometry **nArr = NULL) : size(nSize), arr(nArr) {}
} SortFrame;

__device__ inline void cudaSort(Geometry *list[], int end, int axis) {
   SortFrame stack[kMaxStackSize];
   int stackSize = 0;
   bool stackPopped = false;

   int size = end;
   Geometry **arr = list;
   while (1) {
      if (size == 1) {}
      else if (size == 2) {
         if (arr[0]->getCenter()[axis] < arr[1]->getCenter()[axis]) {
            SWAP(arr[0], arr[1]);
         }
      } else {
         if (!stackPopped) {
            int pivot = size / 2;
            SWAP(arr[pivot], arr[size - 1]);
            int topOfBottom = 0;
            for (int i = 0; i < size - 1; i++) {
               if(arr[i] < arr[size - 1]) {
                  SWAP(arr[i], arr[topOfBottom++]);   
               }             }
            stack[stackSize++] = SortFrame(size, arr); 

         }
      }


      if (stackSize == 0) break;
      arr = stack[stackSize - 1].arr;
      size = stack[stackSize - 1].size;
      stackSize--;
      stackPopped = true;
   }
}

__global__ void sortPieces(Geometry *geomList[], int geomCount, int div, int subDiv, int axis) {
   int idx = blockIdx.x * threadIdx.x;
   int size = subDiv;

   if (idx > div) return;

   if ((subDiv + 1) * idx > geomCount) {
      size = geomCount - subDiv * idx;
      if (size == 0) return;
   }
   cudaSort(geomList + subDiv * idx, subDiv, axis);
}

//crazy stuff
//__global__ void createBVH(Geometry *geomList[], int geomCount, Plane *planeList[], int planeCount, BVHTree *tree) {
//   //Change this back to static memory once I get things working
//   BVHStackEntry stack[kMaxStackSize];
//   tree->root = new BVHNode();
//   tree->planeList = planeList;
//   tree->planeListSize = planeCount;
//
//   
//}
__global__ void createBVH(Geometry *geomList[], int geomCount, Plane *planeList[], int planeCount, BVHTree *tree) {
   //Change this back to static memory once I get things working
   BVHStackEntry stack[kMaxStackSize];
   tree->root = new BVHNode();
   tree->planeList = planeList;
   tree->planeListSize = planeCount;

   BVHNode *cursor = tree->root;
   Geometry **arr = geomList;
   int listSize = geomCount;
   int axis = kXAxis;
   int stackSize = 0;

   // Call the BVHNode constructor

   do {
      if (stackSize >= kMaxStackSize) {
         printf("Stack completely full, aborting");
         return;
      }

      if (listSize == 1) {
         cursor->left = new BVHNode(arr[0]);
         // TODO this is creating a bounding box around a bounding box around 1 item
         cursor->bb = cursor->left->bb;
      } else if (listSize == 2) {
         cursor->left = new BVHNode(arr[0]);
         cursor->right = new BVHNode(arr[1]);
         cursor->bb = combineBoundingBox(cursor->left->bb, cursor->right->bb);
      } else {
         // If the leftside is empty, recursively create that first
         if (!cursor->left) {
            cudaSort(arr, listSize, axis);
            cursor->left = new BVHNode();

            stack[stackSize++] = BVHStackEntry(arr, cursor, listSize, axis);

            cursor = cursor->left;
            listSize = listSize / 2;
            axis = (axis + 1) % kAxisNum;
            continue;
         // Otherwise make the rightside
         } else if (!cursor->right) {
            cursor->right = new BVHNode();

            stack[stackSize++] = BVHStackEntry(arr, cursor, listSize, axis);

            cursor = cursor->right;
            arr = arr + listSize / 2;
            listSize = (listSize - 1) / 2 + 1;
            axis = (axis + 1) % kAxisNum;
            continue;
         } else {
            cursor->bb = combineBoundingBox(cursor->left->bb, cursor->right->bb);
         }

      }

      if (stackSize > 0) {
         // Pop the stack
         cursor = stack[stackSize - 1].cursor;
         listSize = stack[stackSize - 1].listSize;
         arr = stack[stackSize - 1].arr;
         axis = stack[stackSize - 1].axis;
      }
      stackSize--; 
   } while (stackSize >= 0); 
}

__global__ void deleteScene(Geometry *geomList[], int geomCount, Light *lightList[], int lightCount, Shader **shader) {
   // This should really only be run with one thread and block anyways, but this is a safety check
   if (blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0 && threadIdx.y == 0) {
      delete *shader;

      for (int i = 0; i < geomCount; i++) {
         delete geomList[i];
      }

      for (int i = 0; i < lightCount; i++) {
         delete lightList[i];
      }
   }
}

__global__ void rayTrace(int resWidth, int resHeight, Camera cam,
      BVHTree *tree, Light *lights[], int lightCount,  
      vec3 output[], Shader **shader) {

   int x = blockIdx.x * blockDim.x + threadIdx.x;
   int y = blockIdx.y * blockDim.y + threadIdx.y;

   if (x >= resWidth || y >= resHeight)
      return;

   int index = y * resWidth + x;

   // Generate rays
   //Image space coordinates 
   float u = 2.0f * (x / (float)resWidth) - 1.0f; 
   float v = 2.0f * (y / (float)resHeight) - 1.0f;

   // .5f is because the magnitude of cam.right and cam.up should be equal
   // to the width and height of the image plane in world space
   vec3 rPos = u *.5f * cam.right + v * .5f * cam.up + cam.pos;
   vec3 rDir = rPos - cam.pos + cam.lookAtDir;
   Ray ray(rPos, rDir);

   float t;
   Geometry *closestGeom;
   getClosestIntersection(ray, tree, &closestGeom, &t);

   if (closestGeom != kNoShapeFound) {
      vec3 totalColor = shadeObject<kMaxRecurse>(tree, lights, lightCount, 
            closestGeom, t, ray, shader);

      output[index] = vec3(clamp(totalColor.x, 0, 1), 
                           clamp(totalColor.y, 0, 1), 
                           clamp(totalColor.z, 0, 1)); 
   } else {
      output[index] = vec3(0.0f);
   }
}

__global__ void averageBuffer(int resWidth, int resHeight, int sampleCountSqrRoot, uchar4 *output, vec3 *antiAliasBuffer) {
   int x = blockIdx.x * blockDim.x + threadIdx.x;
   int y = blockIdx.y * blockDim.y + threadIdx.y;
   uchar4 clr;
   
   int outputIndex = y * resWidth + x;

   if (x >= resWidth || y >= resHeight)
      return;

   vec3 endColor(0.0f);
   for (int xOffset = 0; xOffset < sampleCountSqrRoot; xOffset++) {
      for (int yOffset = 0; yOffset < sampleCountSqrRoot; yOffset++) {
         int bufferIndex = x * sampleCountSqrRoot + xOffset + (y * sampleCountSqrRoot + yOffset) * resWidth * sampleCountSqrRoot;
         endColor += antiAliasBuffer[bufferIndex];
      }
   }
   endColor /= sampleCountSqrRoot * sampleCountSqrRoot;
   endColor *= 255;

   clr.x = endColor.x; clr.y = endColor.y; clr.z = endColor.z; clr.w = 255;
   output[outputIndex] = clr; 
}

void allocateGPUScene(TKSceneData *data, Geometry ***dGeomList, Plane ***dPlaneList, 
      Light ***dLightList, int *retGeometryCount, int *retPlaneCount,
      int *retLightCount, Shader **dShader, ShadingType stype) {
   int geometryCount = 0;
   int lightCount = 0;

   TKSphere *dSphereTokens = NULL;
   TKPlane *dPlaneTokens = NULL;
   TKPointLight *dPointLightTokens = NULL;
   TKTriangle *dTriangleTokens = NULL;
   TKSmoothTriangle *dSmthTriTokens = NULL;
   TKBox *dBoxTokens = NULL;

   // Cuda memory allocation
   int sphereCount = data->spheres.size();
   if (sphereCount > 0) {
      HANDLE_ERROR(hipMalloc(&dSphereTokens, sizeof(TKSphere) * sphereCount));
      HANDLE_ERROR(hipMemcpy(dSphereTokens, &data->spheres[0], 
               sizeof(TKSphere) * sphereCount, hipMemcpyHostToDevice));
      geometryCount += sphereCount;
   }

   int planeCount = data->planes.size();
   if (planeCount > 0) {
      HANDLE_ERROR(hipMalloc(&dPlaneTokens, sizeof(TKPlane) * planeCount));
      HANDLE_ERROR(hipMemcpy(dPlaneTokens, &data->planes[0],
               sizeof(TKPlane) * planeCount, hipMemcpyHostToDevice));
      *retPlaneCount = planeCount;
   }

   int triangleCount = data->triangles.size();
   if (triangleCount > 0) {
      HANDLE_ERROR(hipMalloc(&dTriangleTokens, sizeof(TKTriangle) * triangleCount));
      HANDLE_ERROR(hipMemcpy(dTriangleTokens, &data->triangles[0], 
               sizeof(TKTriangle) * triangleCount, hipMemcpyHostToDevice));
      geometryCount += triangleCount;
   }

   int boxCount = data->boxes.size();
   if (boxCount > 0) {
      HANDLE_ERROR(hipMalloc(&dBoxTokens, sizeof(TKBox) * boxCount));
      HANDLE_ERROR(hipMemcpy(dBoxTokens, &data->boxes[0],
               sizeof(TKBox) * boxCount, hipMemcpyHostToDevice));
      geometryCount += boxCount;
   }

   int smoothTriangleCount = data->smoothTriangles.size();
   if (smoothTriangleCount > 0) {
      HANDLE_ERROR(hipMalloc(&dSmthTriTokens, sizeof(TKSmoothTriangle) * smoothTriangleCount));
      HANDLE_ERROR(hipMemcpy(dSmthTriTokens, &data->smoothTriangles[0],
               sizeof(TKSmoothTriangle) * smoothTriangleCount, hipMemcpyHostToDevice));
      geometryCount += smoothTriangleCount;
   }

   int pointLightCount = data->pointLights.size();
   if (pointLightCount > 0) {
      HANDLE_ERROR(hipMalloc(&dPointLightTokens, 
               sizeof(TKPointLight) * pointLightCount));
      HANDLE_ERROR(hipMemcpy(dPointLightTokens, &data->pointLights[0],
               sizeof(TKPointLight) * pointLightCount, hipMemcpyHostToDevice));
      lightCount += pointLightCount;
   }

   HANDLE_ERROR(hipMalloc(dGeomList, sizeof(Geometry *) * geometryCount));
   HANDLE_ERROR(hipMalloc(dPlaneList, sizeof(Plane *) * planeCount));
   HANDLE_ERROR(hipMalloc(dLightList, sizeof(Light *) * lightCount));

   // Fill up GeomList and LightList with actual objects on the GPU
   initScene<<<1, 1>>>(*dGeomList, *dPlaneList, *dLightList, dSphereTokens, sphereCount, 
         dPlaneTokens, planeCount, dTriangleTokens, triangleCount, dBoxTokens, boxCount, 
         dSmthTriTokens, smoothTriangleCount, dPointLightTokens, pointLightCount, 
         dShader, stype);

   if (dSphereTokens) HANDLE_ERROR(hipFree(dSphereTokens));
   if (dPlaneTokens) HANDLE_ERROR(hipFree(dPlaneTokens));
   if (dTriangleTokens) HANDLE_ERROR(hipFree(dTriangleTokens));
   if (dSmthTriTokens) HANDLE_ERROR(hipFree(dSmthTriTokens));
   if (dBoxTokens) HANDLE_ERROR(hipFree(dBoxTokens));


   *retGeometryCount = geometryCount;
   *retLightCount = lightCount;
}

void freeGPUScene(Geometry **dGeomList, int geomCount, Light **dLightList, 
      int lightCount, Shader **shader) {
   deleteScene<<<1, 1>>>(dGeomList, geomCount, dLightList, lightCount, shader);

   HANDLE_ERROR(hipFree(dGeomList));
   HANDLE_ERROR(hipFree(dLightList));
   HANDLE_ERROR(hipFree(shader));
}

extern "C" void launch_kernel(TKSceneData *data, ShadingType stype, int width, 
      int height, uchar4 *output, int sampleCount) {
   Geometry **dGeomList; 
   Plane **dPlaneList; 
   Light **dLightList;
   Shader **dShader;

   vec3 *dAntiAliasBuffer;
   uchar4 *dOutput;

   BVHTree *dBvhTree;

   int geometryCount;
   int planeCount;
   int lightCount;

   int sqrSampleCount = sqrt(sampleCount);
   if (sqrSampleCount * sqrSampleCount != sampleCount) {
      printf("Invalid sample count: %d. Sample count for anti aliasing must have an integer square root");
      return;
   }

   TKCamera camTK = *data->camera;
   Camera camera(camTK.pos, camTK.up, camTK.right, 
                 glm::normalize(camTK.lookAt - camTK.pos));

   // Fill the geomList and light list with objects dynamically created on the GPU
   HANDLE_ERROR(hipMalloc(&dShader, sizeof(Shader*)));
   HANDLE_ERROR(hipMalloc(&dOutput, sizeof(uchar4) * width * height));
   HANDLE_ERROR(hipMalloc(&dAntiAliasBuffer, sizeof(vec3) * width * height * sampleCount));
   allocateGPUScene(data, &dGeomList, &dPlaneList, &dLightList, &geometryCount, &planeCount, &lightCount, dShader, stype);
   hipDeviceSynchronize();
   checkCUDAError("AllocateGPUScene failed");

   HANDLE_ERROR(hipMalloc(&dBvhTree, sizeof(BVHTree)));
   createBVH<<<1, 1>>>(dGeomList, geometryCount, dPlaneList, planeCount, dBvhTree);
   hipDeviceSynchronize();
   checkCUDAError("CreateBVH failed");

   // Crazy stuff
   /*int div = 2;
   int subDivs;
   int axis = kXAxis;
   do {
      subDivs = geometryCount / div;
      dim3 dimBlock(kBlockWidth * kBlockWidth);
      dim3 dimGrid((div - 1) / kBlockWidth + 1);
      sortPieces<<<dimBlock, dimGrid>>>(dGeomList, geometryCount, div, subDivs, axis);
      axis = (axis + 1) % kAxisNum;
      
      div *= 2;
      subDivs = geometryCount / div;
   } while ( subDivs > 2);*/

   int antiAliasWidth = width * sqrSampleCount;
   int antiAliasHeight = height * sqrSampleCount;
   dim3 dimBlock(kBlockWidth, kBlockWidth);
   dim3 dimGrid((antiAliasWidth - 1) / kBlockWidth + 1, (antiAliasHeight- 1) / kBlockWidth + 1);
   rayTrace<<<dimGrid, dimBlock>>>(width * sqrSampleCount, height * sqrSampleCount, camera, 
         dBvhTree, dLightList, lightCount, dAntiAliasBuffer, dShader);
   hipDeviceSynchronize();
   checkCUDAError("RayTrace kernel failed");

   dimGrid = dim3((width - 1) / kBlockWidth + 1, (height - 1) / kBlockWidth + 1);
   averageBuffer<<<dimGrid, dimBlock>>>(width, height, sqrSampleCount, dOutput, dAntiAliasBuffer);
   hipDeviceSynchronize();
   checkCUDAError("averageBuffer kernel failed");

   // Clean up
   //freeGPUScene(dGeomList, geometryCount, dLightList, lightCount, dShader);
   HANDLE_ERROR(hipMemcpy(output, dOutput, 
            sizeof(uchar4) * width * height, hipMemcpyDeviceToHost));
   HANDLE_ERROR(hipFree(dOutput));
}

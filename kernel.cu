#include "hip/hip_runtime.h"
#include <stdio.h>
#include <float.h>

#include "Geometry.h"
#include "Light.h"
#include "PointLight.h"
#include "Sphere.h"
#include "Plane.h"
#include "glm/glm.hpp"
#include "kernel.h"
#include "Shader.h"
#include "PhongShader.h"
#include "CookTorranceShader.h"
#include "hipError_t.h"

const int kBlockWidth = 16;
const int kNoShapeFound = -1;
const float kMaxDist = FLT_MAX;

using glm::vec3;

__device__ bool isInShadow(Ray shadow, Geometry *geomList[], int geomCount, 
      int objIdx) {
   float t = FLT_MAX;

   for (int i = 0; i < geomCount; i++) {
      if (i == objIdx) continue;

      float dist = geomList[i]->getIntersection(shadow);

      if (dist > 0.0f) return true;
   }
   return false;
}

// Find the closest shape. The index of the intersecting object is stored in
// retOjIdx and the t-value along the input ray is stored in retParam
//
// If no intersection is found, retObjIdx is set to 'kNoShapeFound'
__device__ void getClosestIntersection(Ray ray, Geometry *geomList[], 
                                       int geomCount, int *retObjIdx, 
                                       float *retParam) {
   float t = kMaxDist;
   int closestShapeIdx = kNoShapeFound;
   for (int i = 0; i < geomCount; i++) {
      float dist = geomList[i]->getIntersection(ray);
      if (dist > 0.0f && dist < t) {
         closestShapeIdx = i;
         t = dist;
      }
   }

   *retObjIdx = closestShapeIdx;
   *retParam = t;
}

//Note: The ray parameter must stay as a copy (not an instance) 
__device__ vec3 shadeObject(Geometry *geomList[], int geomCount, 
                              Light *lights[], int lightCount, int objIdx, 
                              float intParam, Ray ray, Shader **shader) {
      glm::vec3 intersectPoint = ray.getPoint(intParam);
      Material m = geomList[objIdx]->getMaterial();
      vec3 totalLight(0.0f);

      vec3 light, lightDir, normal, eyeVec;
      float compoundedRefl = 1.0f;


      for (int bounce = 0; bounce < kMaxRecurseCount; bounce++) {
         if (bounce > 0) {
            if (m.refl <= 0.0) break;
            compoundedRefl *= m.refl;
         
            vec3 reflect = 2.0f * glm::dot(normal, eyeVec) * normal - eyeVec;
            ray = Ray(intersectPoint, reflect);

            getClosestIntersection(ray, geomList, geomCount, &objIdx, &intParam);
            if (objIdx == kNoShapeFound) break;
            m = geomList[objIdx]->getMaterial();
            intersectPoint = ray.getPoint(intParam);
         }

         for(int lightIdx = 0; lightIdx < lightCount; lightIdx++) {
            light = lights[lightIdx]->getLightAtPoint(geomList, geomCount, objIdx, intersectPoint);
            lightDir = lights[lightIdx]->getLightDir(intersectPoint);
            normal = geomList[objIdx]->getNormalAt(ray, intParam);
            eyeVec = glm::normalize(-ray.d);

            Ray shadow = lights[lightIdx]->getShadow(intersectPoint);
            bool inShadow = isInShadow(shadow, geomList, geomCount, objIdx);

            totalLight += compoundedRefl * (*shader)->shade(m.clr, m.amb, m.dif, 
                  m.spec, m.rough, eyeVec, lightDir, light, normal, inShadow); 

         }
      }

      return totalLight;
}

__global__ void initScene(Geometry *geomList[], Light *lights[], TKSphere *sphereTks, int numSpheres,
      TKPlane *planeTks, int numPlanes, TKPointLight *pLightTks, int numPointLights, 
      Shader **shader, ShadingType stype) {
   int geomIdx = 0;
   int lightIdx = 0;

   // This should really only be run with one thread and block anyways, but this is a safety check
   if (blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0 && threadIdx.y == 0) {

      // Setup the shader
      switch(stype) {
         case PHONG:
           *shader = new PhongShader(); 
           break;
         case COOK_TORRANCE:
           *shader = new CookTorranceShader();
           break;
         default:
           printf("Improper shading type specified\n");
           break;
      }

      // Add all the geometry
      for (int i = 0; i < numSpheres; i++) {
         const TKSphere &s = sphereTks[i];
         const TKFinish f = s.mod.fin;
         Material m(s.mod.pig.clr, f.amb, f.dif, f.spec, f.rough, f.refl, f.refr, f.ior);
         geomList[geomIdx++] = new Sphere(s.p, s.r, m, s.mod.trans, s.mod.invTrans);
      }

      for (int i = 0; i < numPlanes; i++) {
         const TKPlane &p = planeTks[i];
         const TKFinish &f = p.mod.fin;
         Material m(p.mod.pig.clr, f.amb, f.dif, f.spec, f.rough, f.refl, f.refr, f.ior);
         geomList[geomIdx++] = new Plane(p.d, p.n, m, p.mod.trans, p.mod.invTrans);
      }

      // Add all the lights
      for (int i = 0; i < numPointLights; i++) {
         TKPointLight &p = pLightTks[i];
         lights[lightIdx++] = new PointLight(p.pos, p.clr);
      }
   }
}

__global__ void deleteScene(Geometry *geomList[], int geomCount, Light *lightList[], int lightCount) {
   // This should really only be run with one thread and block anyways, but this is a safety check
   if (blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0 && threadIdx.y == 0) {
      for (int i = 0; i < geomCount; i++) {
         delete geomList[i];
      }

      for (int i = 0; i < lightCount; i++) {
         delete lightList[i];
      }
   }
}


__global__ void rayTrace(int resWidth, int resHeight, TKCamera cam,
      Geometry *geomList[], int geomCount, Light *lights[], int lightCount,  
      uchar4 *output, Shader **shader) {

   int x = blockIdx.x * blockDim.x + threadIdx.x;
   int y = blockIdx.y * blockDim.y + threadIdx.y;

   if (x >= resWidth || y >= resHeight)
      return;

   int index = y * resWidth + x;
   uchar4 clr;
   
   // Generate rays
   //Image space coordinates 
   float u = 2.0f * (x / (float)resWidth) - 1.0f; 
   float v = 2.0f * (y / (float)resHeight) - 1.0f;

   // .5f is because the magnitude of cam.right and cam.up should be equal
   // to the width and height of the image plane in world space
   vec3 rPos = u *.5f * cam.right + v * .5f * cam.up + cam.pos;

   //TODO if the cam.lookAt - cam.pos was already normalized, could lead to 
   // speedups
   vec3 lookAtVec = glm::normalize(cam.lookAt - cam.pos);
   vec3 rDir = rPos - cam.pos + lookAtVec;
   Ray r(rPos, rDir);

   float t;
   int closestShapeIdx;
   getClosestIntersection(r, geomList, geomCount, &closestShapeIdx, &t);

   if (closestShapeIdx != kNoShapeFound) {
      vec3 totalColor = shadeObject(geomList, geomCount, lights, lightCount, 
                        closestShapeIdx, t, r, shader);

      clr.x = clamp(totalColor.x * 255.0, 0.0f, 255.0f); 
      clr.y = clamp(totalColor.y * 255.0, 0.0f, 255.0f); 
      clr.z = clamp(totalColor.z * 255.0, 0.0f, 255.0f); 
      clr.w = 255;
   } else {
      clr.x = 0; clr.y = 0; clr.z = 0; clr.w = 255;
   }

   output[index] = clr;
}

void allocateGPUScene(TKSceneData *data, Geometry ***dGeomList, Light ***dLightList, 
   int *retGeometryCount, int *retLightCount, Shader **dShader, ShadingType stype) {
  int geometryCount = 0;
  int lightCount = 0;

  TKSphere *dSphereTokens = NULL;
  TKPlane *dPlaneTokens = NULL;
  TKPointLight *dPointLightTokens = NULL;

  // Cuda memory allocation
  int sphereCount = data->spheres.size();
  if (sphereCount > 0) {
     HANDLE_ERROR(hipMalloc(&dSphereTokens, sizeof(TKSphere) * sphereCount));
     HANDLE_ERROR(hipMemcpy(dSphereTokens, &data->spheres[0], 
              sizeof(TKSphere) * sphereCount, hipMemcpyHostToDevice));
     geometryCount += sphereCount;
  }
  
  int planeCount = data->planes.size();
  if (planeCount > 0) {
     HANDLE_ERROR(hipMalloc(&dPlaneTokens, sizeof(TKPlane) * planeCount));
     HANDLE_ERROR(hipMemcpy(dPlaneTokens, &data->planes[0],
           sizeof(TKPlane) * planeCount, hipMemcpyHostToDevice));
     geometryCount += planeCount;
  }

  int pointLightCount = data->pointLights.size();
  if (pointLightCount > 0) {
      HANDLE_ERROR(hipMalloc(&dPointLightTokens, 
             sizeof(TKPointLight) * pointLightCount));
      HANDLE_ERROR(hipMemcpy(dPointLightTokens, &data->pointLights[0],
            sizeof(TKPointLight) * pointLightCount, hipMemcpyHostToDevice));
      lightCount += pointLightCount;
  }

  HANDLE_ERROR(hipMalloc(dGeomList, sizeof(Geometry *) * geometryCount));
  HANDLE_ERROR(hipMalloc(dLightList, sizeof(Light *) * lightCount));

  // Fill up GeomList and LightList with actual objects on the GPU
  initScene<<<1, 1>>>(*dGeomList, *dLightList, dSphereTokens, sphereCount, dPlaneTokens, 
        planeCount, dPointLightTokens, pointLightCount, dShader, stype);

  if (dSphereTokens) HANDLE_ERROR(hipFree(dSphereTokens));
  if (dPlaneTokens) HANDLE_ERROR(hipFree(dPlaneTokens));

  *retGeometryCount = geometryCount;
  *retLightCount = lightCount;
}

void freeGPUScene(Geometry **dGeomList, int geomCount, Light **dLightList, 
      int lightCount) {
   deleteScene<<<1, 1>>>(dGeomList, geomCount, dLightList, lightCount);

  HANDLE_ERROR(hipFree(dGeomList));
  HANDLE_ERROR(hipFree(dLightList));
}

extern "C" void launch_kernel(TKSceneData *data, ShadingType stype, int width, 
                              int height, uchar4 *output) {
  Geometry **dGeomList; 
  Light **dLightList;
  Shader **dShader;

  uchar4 *dOutput;

  int geometryCount;
  int lightCount;


  HANDLE_ERROR(hipMalloc(&dShader, sizeof(Shader*)));
  HANDLE_ERROR(hipMalloc(&dOutput, sizeof(uchar4) * width * height));

  allocateGPUScene(data, &dGeomList, &dLightList, &geometryCount, &lightCount, dShader, stype);
  hipDeviceSynchronize();
  checkCUDAError("AllocateGPUScene failed");

  dim3 dimBlock(kBlockWidth, kBlockWidth);
  dim3 dimGrid((width - 1) / kBlockWidth + 1, (height - 1) / kBlockWidth + 1);
  rayTrace<<<dimGrid, dimBlock>>>(width, height, *data->camera, 
        dGeomList, geometryCount, dLightList, lightCount, dOutput, dShader);

  hipDeviceSynchronize();
  checkCUDAError("RayTrace kernel failed");

  freeGPUScene(dGeomList, geometryCount, dLightList, lightCount);
  HANDLE_ERROR(hipMemcpy(output, dOutput, 
           sizeof(uchar4) * width * height, hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipFree(dOutput));
}

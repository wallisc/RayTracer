#include "hip/hip_runtime.h"
#include <stdio.h>
#include <float.h>

#include "Light.h"
#include "Camera.h"
#include "PointLight.h"
#include "Sphere.h"
#include "Plane.h"
#include "Triangle.h"
#include "glm/glm.hpp"
#include "PhongShader.h"
#include "SmoothTriangle.h"
#include "CookTorranceShader.h"
#include "hipError_t.h"
#include "kernel.h"

using glm::vec3;

const int kBlockWidth = 16;
const int kNoShapeFound = -1;
const float kMaxDist = FLT_MAX;

__device__ bool isInShadow(const Ray &shadow, Geometry *geomList[], int geomCount, float intersectParam) {
   for (int i = 0; i < geomCount; i++) {
      float dist = geomList[i]->getIntersection(shadow);
      if (isFloatAboveZero(dist) && isFloatLessThan(dist, intersectParam)) { 
         return true;
      }
   }
   return false;
}

// Find the closest shape. The index of the intersecting object is stored in
// retOjIdx and the t-value along the input ray is stored in retParam
//
// If no intersection is found, retObjIdx is set to 'kNoShapeFound'
__device__ void getClosestIntersection(const Ray &ray, Geometry *geomList[], 
      int geomCount, int *retObjIdx, float *retParam) {
   float t = kMaxDist;
   int closestShapeIdx = kNoShapeFound;

   for (int i = 0; i < geomCount; i++) {
      float dist = geomList[i]->getIntersection(ray);

      // If two faces are very close, this picks the face that's normal
      // is closer to the incoming ray
      if (isFloatEqual(t, dist)) {
         glm::vec3 oldNorm = geomList[closestShapeIdx]->getNormalAt(ray, t);
         glm::vec3 newNorm = geomList[i]->getNormalAt(ray, dist);
         glm::vec3 eye = glm::normalize(-ray.d);
         if (glm::dot(eye, newNorm) > glm::dot(eye, oldNorm)) {
            closestShapeIdx = i;
            t = dist;
         }

      // Otherwise, if one face is front of the current one
      } else if (isFloatAboveZero(dist) && dist < t) {
         closestShapeIdx = i;
         t = dist;
      }
   }

   *retObjIdx = closestShapeIdx;
   *retParam = t;
}

//Note: The ray parameter must stay as a copy (not a reference) 
template <int invRecLevel> 
__device__ vec3 shadeObject(Geometry *geomList[], int geomCount, 
      Light *lights[], int lightCount, int objIdx, 
      float intParam, Ray ray, Shader **shader) {

   glm::vec3 intersectPoint = ray.getPoint(intParam);
   Material m = geomList[objIdx]->getMaterial();
   vec3 normal = geomList[objIdx]->getNormalAt(ray, intParam);
   vec3 eyeVec = glm::normalize(-ray.d);
   vec3 totalLight(0.0f);

   for(int lightIdx = 0; lightIdx < lightCount; lightIdx++) {
      vec3 light = lights[lightIdx]->getLightAtPoint(geomList, geomCount, 
                                                     objIdx, intersectPoint);
      vec3 lightDir = lights[lightIdx]->getLightDir(intersectPoint);
      Ray shadow = lights[lightIdx]->getShadowFeeler(intersectPoint);
      float intersectParam = geomList[objIdx]->getIntersection(shadow);
      bool inShadow = isInShadow(shadow, geomList, geomCount, intersectParam); 

      totalLight += (*shader)->shade(m.clr, m.amb, m.dif, m.spec, m.rough, 
            eyeVec, lightDir, light, normal, 
            inShadow);
   }

   vec3 reflectedLight(0.0f);
   if (m.refl > 0.0f && invRecLevel > 0) {

      Ray reflectRay(intersectPoint, 2.0f * glm::dot(normal, eyeVec) * normal - eyeVec);
      int reflObjIdx;
      float reflParam;

      getClosestIntersection(reflectRay, geomList, geomCount, &reflObjIdx, &reflParam);
      if (reflObjIdx != kNoShapeFound) {
         reflectedLight = shadeObject<invRecLevel - 1>(geomList, geomCount, 
               lights, lightCount,
               reflObjIdx, reflParam,
               reflectRay, shader);
      }
   }

   vec3 refractedLight(0.0f);
   if (m.refr > 0.0f && invRecLevel > 0) {
      float n1, n2;
      vec3 refrNorm;
      vec3 d = -eyeVec;

      if (isFloatLessThan(glm::dot(eyeVec, normal), 0.0f)) {
         n1 = m.ior; n2 = kAirIOR;
         refrNorm = -normal;
      } else { 
         n1 = kAirIOR; n2 = m.ior;
         refrNorm = normal;
      }

      float dDotN = glm::dot(d, refrNorm);
      float nr = n1 / n2;
      float discriminant = 1.0f - nr * nr * (1.0f - dDotN * dDotN);
      if (discriminant > 0.0f) {
         vec3 refracDir = nr * (d - refrNorm * dDotN) - refrNorm * sqrt(discriminant);
         Ray refracRay(intersectPoint, refracDir);
         int refrObjIdx;
         float refrParam;
         getClosestIntersection(refracRay, geomList, geomCount, &refrObjIdx, &refrParam);
         if (refrObjIdx != kNoShapeFound) {
            refractedLight = shadeObject<invRecLevel - 1>(geomList, geomCount, 
                  lights, lightCount,
                  refrObjIdx, refrParam,
                  refracRay, shader);
         }
      }

   }

   return totalLight * (1.0f - m.refl - m.refr)
      + m.refl * reflectedLight+ m.refr * refractedLight;
}

template <> 
__device__ vec3 shadeObject<0>(Geometry *geomList[], int geomCount, 
      Light *lights[], int lightCount, int objIdx, 
      float intParam, Ray ray, Shader **shader) { return vec3(0.0f); }

__global__ void initScene(Geometry *geomList[], Light *lights[], TKSphere *sphereTks, int numSpheres,
      TKPlane *planeTks, int numPlanes, TKTriangle *triangleTks, int numTris, 
      TKSmoothTriangle *smthTriTks, int numSmthTris, TKPointLight *pLightTks, int numPointLights, 
      Shader **shader, ShadingType stype) {
   int geomIdx = 0;
   int lightIdx = 0;

   // This should really only be run with one thread and block anyways, but this is a safety check
   if (blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0 && threadIdx.y == 0) {

      // Setup the shader
      switch(stype) {
      case PHONG:
         *shader = new PhongShader(); 
         break;
      case COOK_TORRANCE:
         *shader = new CookTorranceShader();
         break;
      default:
         printf("Improper shading type specified\n");
         break;
      }

      // Add all the geometry
      for (int i = 0; i < numSpheres; i++) {
         const TKSphere &s = sphereTks[i];
         const TKFinish f = s.mod.fin;
         Material m(s.mod.pig.clr, f.amb, f.dif, f.spec, f.rough, f.refl, f.refr, f.ior);
         geomList[geomIdx++] = new Sphere(s.p, s.r, m, s.mod.trans, s.mod.invTrans);
      }

      for (int i = 0; i < numPlanes; i++) {
         const TKPlane &p = planeTks[i];
         const TKFinish &f = p.mod.fin;
         Material m(p.mod.pig.clr, f.amb, f.dif, f.spec, f.rough, f.refl, f.refr, f.ior);
         geomList[geomIdx++] = new Plane(p.d, p.n, m, p.mod.trans, p.mod.invTrans);
      }

      for (int i = 0; i < numTris; i++) {
         const TKTriangle &t = triangleTks[i];
         const TKFinish f = t.mod.fin;
         Material m(t.mod.pig.clr, f.amb, f.dif, f.spec, f.rough, f.refl, f.refr, f.ior);
         geomList[geomIdx++] = new Triangle(t.p1, t.p2, t.p3, m, t.mod.trans, 
               t.mod.invTrans);
      }

      for (int i = 0; i < numSmthTris; i++) {
         const TKSmoothTriangle &t = smthTriTks[i];
         const TKFinish f = t.mod.fin;
         Material m(t.mod.pig.clr, f.amb, f.dif, f.spec, f.rough, f.refl, f.refr, f.ior);
         geomList[geomIdx++] = new SmoothTriangle(t.p1, t.p2, t.p3, t.n1, t.n2, t.n3, 
               m, t.mod.trans, t.mod.invTrans);

      }

      // Add all the lights
      for (int i = 0; i < numPointLights; i++) {
         TKPointLight &p = pLightTks[i];
         lights[lightIdx++] = new PointLight(p.pos, p.clr);
      }
   }
}

__global__ void deleteScene(Geometry *geomList[], int geomCount, Light *lightList[], int lightCount, Shader **shader) {
   // This should really only be run with one thread and block anyways, but this is a safety check
   if (blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0 && threadIdx.y == 0) {
      delete *shader;

      for (int i = 0; i < geomCount; i++) {
         delete geomList[i];
      }

      for (int i = 0; i < lightCount; i++) {
         delete lightList[i];
      }
   }
}

__global__ void rayTrace(int resWidth, int resHeight, Camera cam,
      Geometry *geomList[], int geomCount, Light *lights[], int lightCount,  
      uchar4 *output, Shader **shader, int blockOffset) {

   int blocksPerRow = (resWidth - 1) / blockDim.x + 1;
   int blockX = (blockIdx.x + blockOffset) % blocksPerRow;
   int blockY = (blockIdx.x + blockOffset) / blocksPerRow;
   int x = blockX * blockDim.x + threadIdx.x;
   int y = blockY * blockDim.y + threadIdx.y;

   if (x >= resWidth || y >= resHeight)
      return;

   int index = y * resWidth + x;
   uchar4 clr;

   // Generate rays
   //Image space coordinates 
   float u = 2.0f * (x / (float)resWidth) - 1.0f; 
   float v = 2.0f * (y / (float)resHeight) - 1.0f;

   // .5f is because the magnitude of cam.right and cam.up should be equal
   // to the width and height of the image plane in world space
   vec3 rPos = u *.5f * cam.right + v * .5f * cam.up + cam.pos;
   vec3 rDir = rPos - cam.pos + cam.lookAtDir;
   Ray ray(rPos, rDir);

   float t;
   int closestShapeIdx;
   getClosestIntersection(ray, geomList, geomCount, &closestShapeIdx, &t);

   if (closestShapeIdx != kNoShapeFound) {
      vec3 totalColor = shadeObject<kMaxRecurse>(geomList, geomCount, lights, lightCount, 
            closestShapeIdx, t, ray, shader);

      clr.x = clamp(totalColor.x * 255.0, 0.0f, 255.0f); 
      clr.y = clamp(totalColor.y * 255.0, 0.0f, 255.0f); 
      clr.z = clamp(totalColor.z * 255.0, 0.0f, 255.0f); 
      clr.w = 255;
   } else {
      clr.x = 0; clr.y = 0; clr.z = 0; clr.w = 255;
   }

   output[index] = clr;
}

void allocateGPUScene(TKSceneData *data, Geometry ***dGeomList, Light ***dLightList, 
      int *retGeometryCount, int *retLightCount, Shader **dShader, ShadingType stype) {
   int geometryCount = 0;
   int lightCount = 0;

   TKSphere *dSphereTokens = NULL;
   TKPlane *dPlaneTokens = NULL;
   TKPointLight *dPointLightTokens = NULL;
   TKTriangle *dTriangleTokens = NULL;
   TKSmoothTriangle *dSmthTriTokens = NULL;

   // Cuda memory allocation
   int sphereCount = data->spheres.size();
   if (sphereCount > 0) {
      HANDLE_ERROR(hipMalloc(&dSphereTokens, sizeof(TKSphere) * sphereCount));
      HANDLE_ERROR(hipMemcpy(dSphereTokens, &data->spheres[0], 
               sizeof(TKSphere) * sphereCount, hipMemcpyHostToDevice));
      geometryCount += sphereCount;
   }

   int planeCount = data->planes.size();
   if (planeCount > 0) {
      HANDLE_ERROR(hipMalloc(&dPlaneTokens, sizeof(TKPlane) * planeCount));
      HANDLE_ERROR(hipMemcpy(dPlaneTokens, &data->planes[0],
               sizeof(TKPlane) * planeCount, hipMemcpyHostToDevice));
      geometryCount += planeCount;
   }

   int triangleCount = data->triangles.size();
   if (triangleCount > 0) {
      HANDLE_ERROR(hipMalloc(&dTriangleTokens, sizeof(TKTriangle) * triangleCount));
      HANDLE_ERROR(hipMemcpy(dTriangleTokens, &data->triangles[0], 
               sizeof(TKTriangle) * triangleCount, hipMemcpyHostToDevice));
      geometryCount += triangleCount;
   }

   int smoothTriangleCount = data->smoothTriangles.size();
   if (smoothTriangleCount > 0) {
      HANDLE_ERROR(hipMalloc(&dSmthTriTokens, sizeof(TKSmoothTriangle) * smoothTriangleCount));
      HANDLE_ERROR(hipMemcpy(dSmthTriTokens, &data->smoothTriangles[0],
               sizeof(TKSmoothTriangle) * smoothTriangleCount, hipMemcpyHostToDevice));
      geometryCount += smoothTriangleCount;
   }

   int pointLightCount = data->pointLights.size();
   if (pointLightCount > 0) {
      HANDLE_ERROR(hipMalloc(&dPointLightTokens, 
               sizeof(TKPointLight) * pointLightCount));
      HANDLE_ERROR(hipMemcpy(dPointLightTokens, &data->pointLights[0],
               sizeof(TKPointLight) * pointLightCount, hipMemcpyHostToDevice));
      lightCount += pointLightCount;
   }

   HANDLE_ERROR(hipMalloc(dGeomList, sizeof(Geometry *) * geometryCount));
   HANDLE_ERROR(hipMalloc(dLightList, sizeof(Light *) * lightCount));

   // Fill up GeomList and LightList with actual objects on the GPU
   initScene<<<1, 1>>>(*dGeomList, *dLightList, dSphereTokens, sphereCount, 
         dPlaneTokens, planeCount, dTriangleTokens, triangleCount, 
         dSmthTriTokens, smoothTriangleCount, dPointLightTokens, pointLightCount, 
         dShader, stype);

   if (dSphereTokens) HANDLE_ERROR(hipFree(dSphereTokens));
   if (dPlaneTokens) HANDLE_ERROR(hipFree(dPlaneTokens));
   if (dTriangleTokens) HANDLE_ERROR(hipFree(dTriangleTokens));
   if (dSmthTriTokens) HANDLE_ERROR(hipFree(dSmthTriTokens));

   *retGeometryCount = geometryCount;
   *retLightCount = lightCount;
}

void freeGPUScene(Geometry **dGeomList, int geomCount, Light **dLightList, 
      int lightCount, Shader **shader) {
   deleteScene<<<1, 1>>>(dGeomList, geomCount, dLightList, lightCount, shader);

   HANDLE_ERROR(hipFree(dGeomList));
   HANDLE_ERROR(hipFree(dLightList));
   HANDLE_ERROR(hipFree(shader));
}

extern "C" void launch_kernel(TKSceneData *data, ShadingType stype, int width, 
      int height, uchar4 *output) {
   Geometry **dGeomList; 
   Light **dLightList;
   Shader **dShader;

   uchar4 *dOutput;
   int geometryCount;
   int lightCount;

   TKCamera camTK = *data->camera;
   Camera camera(camTK.pos, camTK.up, camTK.right, 
                 glm::normalize(camTK.lookAt - camTK.pos));

   // Fill the geomList and light list with objects dynamically created on the GPU
   HANDLE_ERROR(hipMalloc(&dShader, sizeof(Shader*)));
   HANDLE_ERROR(hipMalloc(&dOutput, sizeof(uchar4) * width * height));
   allocateGPUScene(data, &dGeomList, &dLightList, &geometryCount, &lightCount, dShader, stype);
   hipDeviceSynchronize();
   checkCUDAError("AllocateGPUScene failed");

   int numKernels = 1;
   int numBlocks = ((width - 1) / kBlockWidth + 1) * ((height - 1) / kBlockWidth + 1);
   int blocksPerKernel = (numBlocks - 1) / numKernels + 1;
   int blockOffset = 0;
   for (int kernelRun = 0; kernelRun < numKernels; kernelRun++) { 
      // Do the actual ray tracing
      dim3 dimBlock(kBlockWidth, kBlockWidth);
      dim3 dimGrid = kernelRun < numKernels - 1 ? dim3(blocksPerKernel)
                     : dim3(numBlocks - (numKernels - 1) * blocksPerKernel);

      rayTrace<<<dimGrid, dimBlock>>>(width, height, camera, 
            dGeomList, geometryCount, dLightList, lightCount, dOutput, dShader, blockOffset);
      hipDeviceSynchronize();
      checkCUDAError("RayTrace kernel failed");
      blockOffset += blocksPerKernel;
   }

   // Clean up
   freeGPUScene(dGeomList, geometryCount, dLightList, lightCount, dShader);
   HANDLE_ERROR(hipMemcpy(output, dOutput, 
            sizeof(uchar4) * width * height, hipMemcpyDeviceToHost));
   HANDLE_ERROR(hipFree(dOutput));
}

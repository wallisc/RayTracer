#include "hip/hip_runtime.h"
#include <stdio.h>
#include <float.h>

#include "Geometry.h"
#include "Light.h"
#include "PointLight.h"
#include "Sphere.h"
#include "Plane.h"
#include "glm/glm.hpp"
#include "kernel.h"
#include "Shader.h"
#include "hipError_t.h"

const int kBlockWidth = 16;
const int kNoShapeFound = -1;
const float kMaxDist = FLT_MAX;

using glm::vec3;

__device__ bool isInShadow(Ray shadow, Geometry *geomList[], int geomCount, 
      int objIdx) {
   float t = FLT_MAX;

   for (int i = 0; i < geomCount; i++) {
      if (i == objIdx) continue;

      float dist = geomList[i]->getIntersection(shadow);

      if (dist > 0.0f) return true;
   }
   return false;
}

__device__ uchar4 shadeObject(Geometry *geomList[], int geomCount, 
                              Light *lights[], int lightCount, int objIdx, 
                              float intParam, Ray ray) {
      uchar4 clr;
      glm::vec3 intersectPoint = ray.getPoint(intParam);
      Material m = geomList[objIdx]->getMaterial();
      vec3 totalLight(0.0f);

      for(int i = 0; i < lightCount; i++) {
         vec3 light = lights[i]->getLightAtPoint(geomList, geomCount, objIdx, intersectPoint);
         vec3 lightDir = lights[i]->getLightDir(intersectPoint);
         vec3 normal = geomList[objIdx]->getNormalAt(ray, intParam);
         Ray shadow = lights[i]->getShadow(intersectPoint);
         totalLight += Shader::shade(m.amb, m.dif, m.spec, m.rough, 
               glm::normalize(-ray.d), lightDir, light, normal,
               isInShadow(shadow, geomList, geomCount, objIdx)); 
      }

      clr.x = clamp(m.clr.x * totalLight.x * 255.0, 0.0f, 255.0f); 
      clr.y = clamp(m.clr.y * totalLight.y * 255.0, 0.0f, 255.0f); 
      clr.z = clamp(m.clr.z * totalLight.z * 255.0, 0.0f, 255.0f); 
      clr.w = 255;
      return clr;
}

__global__ void initScene(Geometry *geomList[], Light *lights[], TKSphere *sphereTks, int numSpheres,
      TKPlane *planeTks, int numPlanes, TKPointLight *pLightTks, int numPointLights) {
   int geomIdx = 0;
   int lightIdx = 0;

   // This should really only be run with one thread and block anyways, but this is a safety check
   if (blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0 && threadIdx.y == 0) {

      // Add all the geometry
      for (int i = 0; i < numSpheres; i++) {
         const TKSphere &s = sphereTks[i];
         const TKFinish f = s.mod.fin;
         Material m(s.mod.pig.clr, f.amb, f.dif, f.spec, f.rough, f.refl, f.refr, f.ior);
         geomList[geomIdx++] = new Sphere(s.p, s.r, m);
      }

      for (int i = 0; i < numPlanes; i++) {
         const TKPlane &p = planeTks[i];
         const TKFinish &f = p.mod.fin;
         Material m(p.mod.pig.clr, f.amb, f.dif, f.spec, f.rough, f.refl, f.refr, f.ior);
         geomList[geomIdx++] = new Plane(p.d, p.n, m);
      }

      // Add all the lights
      for (int i = 0; i < numPointLights; i++) {
         TKPointLight &p = pLightTks[i];
         lights[lightIdx++] = new PointLight(p.pos, p.clr);

      }
   }
}

__global__ void deleteScene(Geometry *geomList[], int geomCount, Light *lightList[], int lightCount) {
   // This should really only be run with one thread and block anyways, but this is a safety check
   if (blockIdx.x == 0 && blockIdx.y == 0 && threadIdx.x == 0 && threadIdx.y == 0) {
      for (int i = 0; i < geomCount; i++) {
         delete geomList[i];
      }

      for (int i = 0; i < lightCount; i++) {
         delete lightList[i];
      }
   }
}

__global__ void rayTrace(int resWidth, int resHeight, TKCamera cam,
      Geometry *geomList[], int geomCount, Light *lights[], int lightCount,  
      uchar4 *output) {

   int x = blockIdx.x * blockDim.x + threadIdx.x;
   int y = blockIdx.y * blockDim.y + threadIdx.y;

   if (x >= resWidth || y >= resHeight)
      return;

   int index = y * resWidth + x;
   uchar4 clr;
   
   // Generate rays
   //Image space coordinates 
   float u = 2.0f * (x / (float)resWidth) - 1.0f; 
   float v = 2.0f * (y / (float)resHeight) - 1.0f;

   //TODO currently makes the assumption that cam.up is normalized
   // .5f is because the magnitude of cam.right and cam.up should be equal
   // to the width and height of the image plane in world space
   vec3 rPos = u *.5f * cam.right + v * .5f * cam.up + cam.pos;

   //TODO if the cam.lookAt - cam.pos was already normalized, could lead to 
   // speedups
   vec3 lookAtVec = glm::normalize(cam.lookAt - cam.pos);
   vec3 rDir = rPos - cam.pos + lookAtVec;
   Ray r(rPos, rDir);

   float t = kMaxDist;
   int closestShapeIdx = kNoShapeFound;
   for (int i = 0; i < geomCount; i++) {
      float dist = geomList[i]->getIntersection(r);
      if (dist > 0.0f && dist < t) {
         closestShapeIdx = i;
         t = dist;
      }
   }

   if (closestShapeIdx != kNoShapeFound) {
      clr = shadeObject(geomList, geomCount, lights, lightCount, 
                        closestShapeIdx, t, r);
   } else {
      clr.x = 0; clr.y = 0; clr.z = 0; clr.w = 255;
   }

   output[index] = clr;
}

void allocateGPUScene(TKSceneData *data, Geometry ***dGeomList, Light ***dLightList, 
   int *retGeometryCount, int *retLightCount) {
  int geometryCount = 0;
  int lightCount = 0;

  TKSphere *dSphereTokens = NULL;
  TKPlane *dPlaneTokens = NULL;
  TKPointLight *dPointLightTokens = NULL;

  // Cuda memory allocation
  int sphereCount = data->spheres.size();
  if (sphereCount > 0) {
     HANDLE_ERROR(hipMalloc(&dSphereTokens, sizeof(TKSphere) * sphereCount));
     HANDLE_ERROR(hipMemcpy(dSphereTokens, &data->spheres[0], 
              sizeof(TKSphere) * sphereCount, hipMemcpyHostToDevice));
     geometryCount += sphereCount;
  }
  
  int planeCount = data->planes.size();
  if (planeCount > 0) {
     HANDLE_ERROR(hipMalloc(&dPlaneTokens, sizeof(TKPlane) * planeCount));
     HANDLE_ERROR(hipMemcpy(dPlaneTokens, &data->planes[0],
           sizeof(TKPlane) * planeCount, hipMemcpyHostToDevice));
     geometryCount += planeCount;
  }

  int pointLightCount = data->pointLights.size();
  if (pointLightCount > 0) {
      HANDLE_ERROR(hipMalloc(&dPointLightTokens, 
             sizeof(TKPointLight) * pointLightCount));
      HANDLE_ERROR(hipMemcpy(dPointLightTokens, &data->pointLights[0],
            sizeof(TKPointLight) * pointLightCount, hipMemcpyHostToDevice));
      lightCount += pointLightCount;
  }

  HANDLE_ERROR(hipMalloc(dGeomList, sizeof(Geometry *) * geometryCount));
  HANDLE_ERROR(hipMalloc(dLightList, sizeof(Light *) * lightCount));

  // Fill up GeomList and LightList with actual objects on the GPU
  initScene<<<1, 1>>>(*dGeomList, *dLightList, dSphereTokens, sphereCount, dPlaneTokens, 
        planeCount, dPointLightTokens, pointLightCount);

  if (dSphereTokens) HANDLE_ERROR(hipFree(dSphereTokens));
  if (dPlaneTokens) HANDLE_ERROR(hipFree(dPlaneTokens));

  *retGeometryCount = geometryCount;
  *retLightCount = lightCount;
}

void freeGPUScene(Geometry **dGeomList, int geomCount, Light **dLightList, 
      int lightCount) {
   deleteScene<<<1, 1>>>(dGeomList, geomCount, dLightList, lightCount);

  HANDLE_ERROR(hipFree(dGeomList));
  HANDLE_ERROR(hipFree(dLightList));
}

extern "C" void launch_kernel(TKSceneData *data, int width, int height, uchar4 *output) {
  Geometry **dGeomList; 
  Light **dLightList;

  uchar4 *dOutput;

  int geometryCount;
  int lightCount;


  allocateGPUScene(data, &dGeomList, &dLightList, &geometryCount, &lightCount);
  HANDLE_ERROR(hipMalloc(&dOutput, sizeof(uchar4) * width * height));


  dim3 dimBlock(kBlockWidth, kBlockWidth);
  dim3 dimGrid((width - 1) / kBlockWidth + 1, (height - 1) / kBlockWidth + 1);
  rayTrace<<<dimGrid, dimBlock>>>(width, height, *data->camera, 
        dGeomList, geometryCount, dLightList, lightCount, dOutput);

  hipDeviceSynchronize();
  checkCUDAError("kernel failed");

  freeGPUScene(dGeomList, geometryCount, dLightList, lightCount);
  HANDLE_ERROR(hipMemcpy(output, dOutput, 
           sizeof(uchar4) * width * height, hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipFree(dOutput));
}



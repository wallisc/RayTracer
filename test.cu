#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "hipError_t.h" 

__global__ void test() {
   printf("Hi!\n");
}

int main() {
   hipStream_t stream1, stream2;
   hipStreamCreate(&stream1);
   hipStreamCreate(&stream2);
   test<<<1, 1, 0, stream1>>>();
   test<<<1, 1, 0, stream2>>>();
   hipDeviceSynchronize();
   checkCUDAError("test failed");
}
